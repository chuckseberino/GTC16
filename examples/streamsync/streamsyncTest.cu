//-----------------------------------------------------------------------------
// Copyright 2016 Chuck Seberino
//
// This file is part of CCT.
//
// CCT is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// CCT is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with CCT.  If not, see <http://www.gnu.org/licenses/>.
//-----------------------------------------------------------------------------
#include "streamsyncTest.hpp"

using namespace CCT;

int StreamsyncTest::numStreams = 4;
int StreamsyncTest::numJobs = 20;
bool StreamsyncTest::syncStreams = true;

//-----------------------------------------------------------------------------
StreamsyncTest::StreamsyncTest(int index)
    : TestClass(index)
{
}

//-----------------------------------------------------------------------------
void StreamsyncTest::run()
{
    // Perform copy and compute test
    gpuPtr->timerStart(EventStream);
    int numIterations = IDIVUP(numJobs, numStreams);
    for (int n = 0; n < numIterations; ++n)
    {
        // Perform initial work as separate streams
        for (int ii = 0; ii < numStreams; ++ii)
        {
            // Wait for previous loop main stream
            gpuPtr->streamWait(ii, EventStream);
            // "Compute"
            gpuPtr->sleep(10+ii*50, ii);
            // Create event record for stream ii
            gpuPtr->timerStop(ii);

            // Break synchronization on last stream 
            if (syncStreams || ii != 3) 
            {
                // Tell main stream to wait for stream ii stop record.
                gpuPtr->streamWait(EventStream, ii);
            }
        }

        // Main stream "Compute"
        gpuPtr->sleep(100, EventStream);
        // Synchronization point for other streams
        gpuPtr->timerStop(EventStream);

        // Perform additional work as individual streams
        for (int ii = 0; ii < numStreams; ++ii)
        {
            // Wait for main stream to be complete.
            gpuPtr->streamWait(ii, EventStream);
            // "Compute"
            gpuPtr->sleep(30+10*ii, ii);
            // Create event record for stream ii
            gpuPtr->timerStop(ii);
            // Tell main stream to wait for stream ii stop record.
            gpuPtr->streamWait(EventStream, ii);
        }

        // Again, consolidate and run on a single stream
        gpuPtr->sleep(100, EventStream);
        // Synchronization point for other streams
        gpuPtr->timerStop(EventStream);
    }

    // Flush all outstanding work to ensure valid timing value.
    gpuPtr->timerStop(EventStream);
    gpuPtr->deviceSynchronize();
    CCT_INFO("Total time = " << gpuPtr->timerElapsed(EventStream) << "ms");
}
